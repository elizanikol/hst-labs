#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <vector>
#include <cmath>
#include <iostream>
#include <numeric>

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "kernel.h"
//#include "helper_functions.h"
//#include "hip/hip_runtime_api.h"


const auto NumThreads = 1024;
const auto SecondsMultiplier = 0.001;


void gpuAssert(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cout << "Cuda error: " << hipGetErrorString(code) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void MaxElement(double* data, int* offsets, double* results, int resultsSize)
{
    const auto threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId >= resultsSize)
    {
        return;
    }

    double maxElement = data[offsets[threadId]];
    const auto rowSize = offsets[threadId + 1] - offsets[threadId];
    for (auto i = 1; i < rowSize; ++i)
    {
        auto tmp = data[offsets[threadId] + i];
        if (tmp > maxElement)
        {
            maxElement = tmp;
        }
    }
    
    results[threadId] = pow(maxElement, 2.0);
}


void flattenMatricesAndCalculateOffsets(double* data, const std::vector<std::vector<double>>& matrices, int* offsets)
{
    double* currentData = data;

    offsets[0] = 0;
    int* currentOffsets = offsets + 1;

    for (size_t i = 0; i < matrices.size(); ++i)
    {
        std::copy(matrices[i].begin(), matrices[i].end(), currentData);
        currentData += matrices[i].size();

        // calculate offsets
        const auto rowSize = static_cast<int>(std::sqrt(matrices[i].size()));
        std::vector<int> rowOffsets(rowSize, rowSize);
        rowOffsets[0] += *(currentOffsets - 1);
        std::partial_sum(rowOffsets.begin(), rowOffsets.end(), currentOffsets);
        currentOffsets += rowSize;
    }
}

void ArrayTo2DVector(double* results, std::vector<std::vector<double>>& maxSquaresVectors)
{
    double* current = results;
    for (auto& v : maxSquaresVectors)
    {
        std::copy(current, current + v.size(), v.begin());
        current += v.size();
    }
}

void prepareData(size_t& dataSize, size_t& offsetsSize, size_t& resultsSize,
    const std::vector<std::vector<double>>& matrices, std::vector<std::vector<double>>& maxSquaresVectors)
{
    for (size_t matrixId = 0; matrixId < matrices.size(); ++matrixId)
    {
        const auto matrixSize = matrices[matrixId].size();
        const auto rowSize = static_cast<int>(std::sqrt(matrixSize));

        dataSize += matrixSize;
        resultsSize += rowSize;

        maxSquaresVectors[matrixId].resize(rowSize);
    }
    offsetsSize = resultsSize + 1;
}

double CalculateMaxElementsSquare(const std::vector<std::vector<double>>& matrices, std::vector<std::vector<double>>& maxSquaresVectors)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t dataSize = 0;
    size_t offsetsSize = 0;
    size_t resultsSize = 0;
    prepareData(dataSize, offsetsSize, resultsSize, matrices, maxSquaresVectors);

    double* data = (double*)malloc(sizeof(double) * dataSize);
    int* offsets = (int*)malloc(sizeof(int) * offsetsSize);
    double* results = (double*)malloc(sizeof(double) * resultsSize);

    double* deviceData;
    int* deviceOffsets;
    double* deviceResults;
    gpuAssert(hipMalloc((void**)&deviceData, sizeof(double) * dataSize));
    gpuAssert(hipMalloc((void**)&deviceOffsets, sizeof(int) * offsetsSize));
    gpuAssert(hipMalloc((void**)&deviceResults, sizeof(double) * resultsSize));

    flattenMatricesAndCalculateOffsets(data, matrices, offsets);

    hipMemcpy(deviceData, data, sizeof(double) * dataSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceOffsets, offsets, sizeof(int) * offsetsSize, hipMemcpyHostToDevice);
    hipMemcpy(deviceResults, results, sizeof(double) * resultsSize, hipMemcpyHostToDevice);

    const auto gridSize = static_cast<int>((resultsSize + NumThreads - 1) / NumThreads);
    const auto blockSize = static_cast<int>(NumThreads);

   /* for (int i = 0; i < dataSize; ++i)
    {
        std::cout << data[i] << ' ';
    }
    std::cout << std::endl;

    for (int i = 0; i < offsetsSize; ++i)
    {
        std::cout << offsets[i] << ' ';
    }
    std::cout << std::endl;*/

    hipEventRecord(start);
    MaxElement<<<gridSize, blockSize>>>(deviceData, deviceOffsets, deviceResults, static_cast<int>(resultsSize));
    //hipDeviceSynchronize();
    hipEventRecord(stop);
    gpuAssert(hipEventSynchronize(stop));

    gpuAssert(hipMemcpy(results, deviceResults, sizeof(double) * resultsSize, hipMemcpyDeviceToHost));

    /*for (int i = 0; i < resultsSize; ++i)
    {
        std::cout << results[i] << ' ';
    }
    std::cout << std::endl;*/
    ArrayTo2DVector(results, maxSquaresVectors);
    gpuAssert(hipFree(deviceData));
    gpuAssert(hipFree(deviceResults));
    gpuAssert(hipFree(deviceOffsets));

    free(data);
    free(offsets);
    free(results);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    return milliseconds * SecondsMultiplier;
}
